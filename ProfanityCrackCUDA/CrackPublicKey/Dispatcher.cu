#include "hip/hip_runtime.h"
﻿
/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V1.3.0
  * @date		28-April-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
// Includes
#include <stdexcept>
#include <iostream>
#include <thread>
#include <sstream>
#include <iomanip>
#include <random>
#include <thread>
#include <algorithm>

#include "hip/hip_runtime.h"
#include ""

#include "precomp.hpp"
#include "Dispatcher.hpp"
#include "Data/Find.h"
#include <GPU.h>


//(any random 256-bit number from 0x1 to 0xFFFF FFFF FFFF FFFF FFFF FFFF FFFF FFFE BAAE DCE6 AF48 A03B BFD2 5E8C D036 4140)
uint64_t* Dispatcher::Device::genPrivateKeys(private_key* priv_keys, size_t size, uint64_t* init_value) {

	uint64_t value[4];
	value[0] = init_value[0];
	value[1] = init_value[1];
	value[2] = init_value[2];
	value[3] = init_value[3];

	for (size_t i = 0; i < size; i++)
	{
		priv_keys[i].key[0] = value[0];
		priv_keys[i].key[1] = value[1];
		priv_keys[i].key[2] = value[2];
		priv_keys[i].key[3] = value[3];
		tools::incUlong4(value);
	}
	return value;
}


Dispatcher::Device::Device(
	Dispatcher& parent,
	DataClass& Data,
	point public_key) :
	m_parent(parent),
	data(Data),
	public_key(public_key),
	m_round(0),
	m_speed(SPEED_SAMPLES)
{

}

Dispatcher::Device::~Device() {

}
Dispatcher::Dispatcher(DataClass& data, ConfigClass& config) :
	data(data),
	config(config)
{

}

Dispatcher::~Dispatcher() {

}

void Dispatcher::addDevice(DataClass& data, point public_key) {
	Device* pDevice = new Device(*this, data, public_key);
	Dev = pDevice;
}


int Dispatcher::cudaMallocDevice(uint8_t** point, uint64_t size, uint64_t* all_gpu_memory_size, std::string buff_name) {
	if (hipMalloc(point, size) != hipSuccess) {
		fprintf(stderr, "hipMalloc (%s) failed! Size: %s", buff_name.c_str(), tools::formatWithCommas(size).data()); return -1;
	}
	*all_gpu_memory_size += size;
	//if(size == 0)
	//	std::cout << "!!! WORNING !!! MALLOC GPU MEMORY SIZE (" << buff_name << "): 0.000000 MB\n";
	//else
	//	std::cout << "MALLOC GPU MEMORY SIZE (" << buff_name << "): " << std::to_string((float)size / (1024.0f * 1024.0f)) << " MB\r";
	return 0;
}

int Dispatcher::memsetGlobal(int mode)
{
	if (hipMemcpyAsync(this->data.dev.public_key, &this->data.host.public_key, sizeof(point), hipMemcpyHostToDevice, this->data.stream1) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.public_key failed!"); return -1; }
	if (hipMemcpyAsync(this->data.dev.precomp, g_precomp, sizeof(g_precomp), hipMemcpyHostToDevice, this->data.stream1) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.precomp failed!"); return -1; }
	if((mode == MODE_SEARCH_IN_GPU_AND_CPU) || (mode == MODE_SEARCH_ONLY_IN_GPU))
		if (hipMemsetAsync(this->data.dev.result, 0, sizeof(resultFound), this->data.stream1) != hipSuccess) { fprintf(stderr, "hipMemset Board->dev.ret failed!"); return -1; }
	return 0;
}

int Dispatcher::memsetSearchGPU() {
	std::cout << "Initialization..." << std::endl;
	std::cout << "May take several minutes..." << std::endl;
	size_t memory_size = 0;
	for (int i = 0; i < 256; i++)
	{
		std::string name = "Table " + tools::byteToHexString(i);
		if (cudaMallocDevice((uint8_t**)&this->data.dev.tables[i].table, this->data.host.tables_gpu[i].size, &memory_size, name.c_str()) != 0)
		{
			std::cout << "Error cudaMallocDevice(), Board->dev.table[i]! i = " << i << std::endl;
			return -1;
		}
		this->data.dev.tables[i].size = this->data.host.tables_gpu[i].size;
		this->data.dev.memory_size += this->data.host.tables_gpu[i].size;
	}
	//std::cout << "MALLOC MEMORY SIZE (TABLES GPU): " << std::to_string((float)memory_size / (1024.0f * 1024.0f)) << " MB\n";

	std::cout << "INIT GPU ... \n";
	for (int i = 0; i < 256; i++)
	{
		if (hipMemcpy((void*)this->data.dev.tables[i].table, this->data.host.tables_gpu[i].table, this->data.host.tables_gpu[i].size, hipMemcpyHostToDevice) != hipSuccess)
		{
			std::cout << "hipMemcpy to Board->dev.table[i] failed! i = " << i << std::endl;
			return -1;
		}
		const size_t percentDone = (i * 100 / 255);
		std::cout << "  " << percentDone << "%\r";
	}
	if (hipMemcpy(this->data.dev.dev_tables, this->data.dev.tables, 256 * sizeof(tableStruct), hipMemcpyHostToDevice) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.table failed!"); return -1; }
	
	if (deviceSynchronize("memsetGPU") != hipSuccess) return -1;

	this->data.host.freeTableBuffersGPU();

	return 0;
}

int Dispatcher::init(int mode) {
	memsetGlobal(mode);
	std::cout << std::endl;
	//dev_crack_init << <(uint32_t)this->config.cuda_grid, (uint32_t)this->config.cuda_block, 0, this->data.stream1 >> > (this->data.dev.precomp, this->data.dev.extension_public_key, this->data.dev.public_key);
	dev_crack_init << <(uint32_t)this->config.cuda_grid, (uint32_t)this->config.cuda_block, 0, this->data.stream1 >> > (this->data.dev.precomp, this->data.dev.pDeltaX, this->data.dev.pPrevLambda, this->data.dev.public_key);
	if (deviceSynchronize("crack_init") != hipSuccess) return -1; //


	return 0;
}

int Dispatcher::startSearchOnlyCPU() {
	uint32_t cuda_grid = (uint32_t)this->config.cuda_grid / PROFANITY_INVERSE_SIZE;
	profanity_inverse << <(uint32_t)cuda_grid, (uint32_t)this->config.cuda_block, 0, this->data.stream1 >> > (this->data.dev.pDeltaX, this->data.dev.pInverse);
	dev_crack_search_only_cpu << <(uint32_t)this->config.cuda_grid, (uint32_t)this->config.cuda_block, 0, this->data.stream1 >> > (
		this->data.dev.pDeltaX,
		this->data.dev.pInverse,
		this->data.dev.pPrevLambda,
		this->data.dev.result_keys);

	hipError_t cudaStatus = hipSuccess;
	if (deviceSynchronize("crack") != hipSuccess) return -1; //
	return 0;
}


int Dispatcher::endSearchOnlyCPU() {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(this->data.host.result_keys, this->data.dev.result_keys, this->data.host.size_result_keys, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy result failed!");
		return -1;
	}
	return 0;
}

int Dispatcher::startSearchOnlyGPU() {

	uint32_t cuda_grid = (uint32_t)this->config.cuda_grid / PROFANITY_INVERSE_SIZE;
	profanity_inverse <<<(uint32_t)cuda_grid, (uint32_t)this->config.cuda_block, 0, this->data.stream1>>>(this->data.dev.pDeltaX, this->data.dev.pInverse);
	dev_crack_search_only_gpu << <(uint32_t)this->config.cuda_grid, (uint32_t)this->config.cuda_block, 0, this->data.stream1 >> >(
		this->data.dev.pDeltaX,
		this->data.dev.pInverse,
		this->data.dev.pPrevLambda,
		this->data.dev.dev_tables,
		this->data.dev.result);
	
	if (deviceSynchronize("crack") != hipSuccess) return -1; //
	return 0;
}


int Dispatcher::endSearchOnlyGPU() {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(&this->data.host.result, this->data.dev.result, sizeof(resultFound), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy result failed!");
		return -1;
	}
	return 0;
}

int Dispatcher::startSearchGPUAndCPU() {

	uint32_t cuda_grid = (uint32_t)this->config.cuda_grid / PROFANITY_INVERSE_SIZE;
	profanity_inverse << <(uint32_t)cuda_grid, (uint32_t)this->config.cuda_block, 0, this->data.stream1 >> > (this->data.dev.pDeltaX, this->data.dev.pInverse);
	dev_crack_search_gpu_cpu << <(uint32_t)this->config.cuda_grid, (uint32_t)this->config.cuda_block, 0, this->data.stream1 >> > (
		this->data.dev.pDeltaX,
		this->data.dev.pInverse,
		this->data.dev.pPrevLambda,
		this->data.dev.dev_tables,
		this->data.dev.result_keys,
		this->data.dev.result);

	if (deviceSynchronize("crack") != hipSuccess) return -1; //
	return 0;
}


int Dispatcher::endSearchGPUAndCPU() {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(&this->data.host.result, this->data.dev.result, sizeof(resultFound), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy result failed!");
		return -1;
	}
	cudaStatus = hipMemcpy(this->data.host.result_keys, this->data.dev.result_keys, this->data.host.size_result_keys, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy result_keys failed!");
		return -1;
	}
	return 0;
}





void Dispatcher::dispatch(Device& d) {

}

void Dispatcher::handleResult() {
	//point* res = Dev->result;
	//if (Search_Keys_In_Memory(Dev->data, res, m_size, Dev->m_round, config) == 1) {
	//	m_quit = true;
	//}
}



// This is run when m_mutex is held.
void Dispatcher::printSpeed(size_t round) {
	std::string strGPUs;
	double speedTotal = 0;

	const auto curSpeed = Dev->m_speed.getSpeed();
	speedTotal += curSpeed;
	strGPUs += " GPU: " + formatSpeed(curSpeed);


	const std::string strVT100ClearLine = "\33[2K\r";
	std::cerr << strVT100ClearLine << "Total: " << formatSpeed(speedTotal) << " -" << strGPUs << ", Round: " << round << '\r' << std::flush;
}


std::string Dispatcher::formatSpeed(double f) {
	const std::string S = " KMGT";

	unsigned int index = 0;
	while (f > 1000.0f && index < S.size()) {
		f /= 1000.0f;
		++index;
	}

	std::ostringstream ss;
	ss << std::fixed << std::setprecision(5) << (double)f << " " << S[index] << "H/s";
	return ss.str();
}
