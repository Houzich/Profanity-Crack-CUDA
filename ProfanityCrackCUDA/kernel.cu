﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V1.3.0
  * @date		28-April-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "CrackPublicKey/Crack.hpp"
#include "Helper.h"

#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "../config/Config.hpp"

int main()
{
	int ret = 0;

	DataClass* Data = new DataClass();
	std::vector <std::string> publicKeys;

	setlocale(LC_ALL, "Russian");
	system("chcp 1251");

	hipError_t cudaStatus = hipSuccess;
	ConfigClass Config;
	try {
		parse_config(&Config, "config.cfg");
	}
	catch (...) {
		for (;;)
			std::this_thread::sleep_for(std::chrono::seconds(30));
	}

	devicesInfo();

	uint32_t num_device = 0;
	int mode = -1;
	int err = -1;
	// address 0x77cc6699448b8c5e9c503e749a16b8374015c976  private_key 0xc8505c6c876399185b499f3c1ae43e5b553496e135dbcc2ca67c4b278cd9bb18
	//std::string pubKeyIn = "7cefe04ddbdb17e3861ec995d515bac16cc2766cca1d66c27acdcee876fb3cd2d811c410835d71c56fab7e492084a3949aa6797aefb38ab4b1ab1dd1b6e15f45";

#ifndef TEST_MODE
	std::cout << "\n\nEnter number of device: ";
	std::cin >> num_device;
#endif //TEST_MODE
	cudaStatus = hipSetDevice(num_device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
#ifndef TEST_MODE
	//std::cout << "Enter the public key you are looking for: ";
	//std::cin >> pubKeyIn;
	//if (pubKeyIn.size() != 128) {
	//	std::cout << "ERROR: incorrect public key length: " << "  [" << pubKeyIn.size() << "]\n";
	//	goto Error;
	//}
#endif //TEST_MODE

	err = tools::readPublicKeys(Config.file_public_keys, publicKeys);
	if (err == -1) {
		std::cout << "Error readPublicKeys!" << std::endl;
		goto Error;
	}

	std::cout << "READ TABLES FOR GPU! WAIT..." << std::endl;
	err = tools::readTables(Data->host.tables_gpu, Config.folder_8_bytes_keys_gpu);
	if (err == -1) {
		std::cerr << "Error readAllTables!" << std::endl;
		goto Error;
	}
	std::cout << "\nREAD TABLES FOR CPU! WAIT..." << std::endl;
	err = tools::readTables(Data->host.tables_cpu, Config.folder_8_bytes_keys_cpu);
	if (err == -1) {
		std::cerr << "Error readAllTables!" << std::endl;
		goto Error;
	}
	Data->host.calcSizeTableBuffersGPU();
	Data->host.calcSizeTableBuffersCPU();
	std::cout << "\nSIZE DATABASE FOR CPU: " << std::to_string((float)(Data->host.size_tables_cpu) / (1024.0f * 1024.0f * 1024.0f)) << " GB\n";
	std::cout << "SIZE DATABASE FOR GPU: " << std::to_string((float)(Data->host.size_tables_gpu) / (1024.0f * 1024.0f * 1024.0f)) << " GB\n";


	if ((Data->host.size_tables_cpu != 0) && (Data->host.size_tables_gpu != 0))
	{
		std::cout << "PROGRAM STARTS IN MODE \"SEARCH IN GPU AND CPU\" \n";
		mode = MODE_SEARCH_IN_GPU_AND_CPU;
	}
	else if ((Data->host.size_tables_cpu != 0) && (Data->host.size_tables_gpu == 0))
	{
		std::cout << "PROGRAM STARTS IN MODE \"SEARCH ONLY IN CPU\" \n";
		mode = MODE_SEARCH_ONLY_IN_CPU;
	}
	else if ((Data->host.size_tables_cpu == 0) && (Data->host.size_tables_gpu != 0))
	{
		std::cout << "PROGRAM STARTS IN MODE \"SEARCH ONLY IN GPU\" \n";
		mode = MODE_SEARCH_ONLY_IN_GPU;
	}
	else if ((Data->host.size_tables_cpu == 0) && (Data->host.size_tables_gpu == 0))
	{
		std::cout << "ERROR TABLES SIZE! SIZE = 0. check the paths in the file confid.cfd \n";
		return -1;
	}

	if (Data->malloc(Config.cuda_grid, Config.cuda_block, mode, false) != 0) {
		std::cerr << "Error Data->Malloc()!" << std::endl;
		goto Error;
	}

	std::cout << "\n*************** Crack START! ********************\n" << std::endl;
	err = crack_public_key(*Data, publicKeys, Config);
	if (err < 0) {
		std::cerr << "Error crack_public_key)!" << std::endl;
		for (;;)
			std::this_thread::sleep_for(std::chrono::seconds(30));
	}

	std::cout << "\n*************** Crack END! **********************\n" << std::endl;
	std::cout << "\n\n";
	std::cout << "FINISH!!!!!\n";
	std::cout << "FINISH!!!!!\n";
	std::cout << "FINISH!!!!!\n";

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return 0;
Error:
	std::cout << "\n\nERROR!" << std::endl;

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return -1;
}

