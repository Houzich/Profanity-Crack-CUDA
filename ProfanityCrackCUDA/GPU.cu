#include "hip/hip_runtime.h"
﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V1.3.0
  * @date		28-April-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
#include <stdafx.h>
#include <stdio.h>


#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <GPU.h>

  // TODO: Add OpenCL kernel code here.
 #define printf(...)


#define MP_NWORDS 8

typedef uint32_t uint32_t;

typedef union {
	uint8_t b[200];
	uint64_t q[25];
	uint32_t d[50];
} ethhash;

__device__
static uint64_t rotate64(uint64_t x, uint32_t i)
{
	uint64_t a = x << i;
	uint64_t b = x >> (64 - i);
	return a | b;
}

//__device__
//static uint32_t rotate32(uint32_t x, uint32_t i)
//{
//	uint32_t a = x << i;
//	uint32_t b = x >> (32 - i);
//	return a | b;
//}

__device__
static uint32_t mul_hi(uint32_t a, uint32_t b)
{
	uint64_t res = (uint64_t)a * (uint64_t)b;
	res = res >> 32;
	return (uint32_t)res;
}

#define TH_ELT(t, c0, c1, c2, c3, c4, d0, d1, d2, d3, d4) \
{ \
    t = rotate64((uint64_t)(d0 ^ d1 ^ d2 ^ d3 ^ d4), (uint64_t)1) ^ (c0 ^ c1 ^ c2 ^ c3 ^ c4); \
}

#define THETA(s00, s01, s02, s03, s04, \
              s10, s11, s12, s13, s14, \
              s20, s21, s22, s23, s24, \
              s30, s31, s32, s33, s34, \
              s40, s41, s42, s43, s44) \
{ \
    TH_ELT(t0, s40, s41, s42, s43, s44, s10, s11, s12, s13, s14); \
    TH_ELT(t1, s00, s01, s02, s03, s04, s20, s21, s22, s23, s24); \
    TH_ELT(t2, s10, s11, s12, s13, s14, s30, s31, s32, s33, s34); \
    TH_ELT(t3, s20, s21, s22, s23, s24, s40, s41, s42, s43, s44); \
    TH_ELT(t4, s30, s31, s32, s33, s34, s00, s01, s02, s03, s04); \
    s00 ^= t0; s01 ^= t0; s02 ^= t0; s03 ^= t0; s04 ^= t0; \
    s10 ^= t1; s11 ^= t1; s12 ^= t1; s13 ^= t1; s14 ^= t1; \
    s20 ^= t2; s21 ^= t2; s22 ^= t2; s23 ^= t2; s24 ^= t2; \
    s30 ^= t3; s31 ^= t3; s32 ^= t3; s33 ^= t3; s34 ^= t3; \
    s40 ^= t4; s41 ^= t4; s42 ^= t4; s43 ^= t4; s44 ^= t4; \
}

#define RHOPI(s00, s01, s02, s03, s04, \
              s10, s11, s12, s13, s14, \
              s20, s21, s22, s23, s24, \
              s30, s31, s32, s33, s34, \
              s40, s41, s42, s43, s44) \
{ \
	t0  = rotate64(s10, (uint64_t) 1);  \
	s10 = rotate64(s11, (uint64_t)44); \
	s11 = rotate64(s41, (uint64_t)20); \
	s41 = rotate64(s24, (uint64_t)61); \
	s24 = rotate64(s42, (uint64_t)39); \
	s42 = rotate64(s04, (uint64_t)18); \
	s04 = rotate64(s20, (uint64_t)62); \
	s20 = rotate64(s22, (uint64_t)43); \
	s22 = rotate64(s32, (uint64_t)25); \
	s32 = rotate64(s43, (uint64_t) 8); \
	s43 = rotate64(s34, (uint64_t)56); \
	s34 = rotate64(s03, (uint64_t)41); \
	s03 = rotate64(s40, (uint64_t)27); \
	s40 = rotate64(s44, (uint64_t)14); \
	s44 = rotate64(s14, (uint64_t) 2); \
	s14 = rotate64(s31, (uint64_t)55); \
	s31 = rotate64(s13, (uint64_t)45); \
	s13 = rotate64(s01, (uint64_t)36); \
	s01 = rotate64(s30, (uint64_t)28); \
	s30 = rotate64(s33, (uint64_t)21); \
	s33 = rotate64(s23, (uint64_t)15); \
	s23 = rotate64(s12, (uint64_t)10); \
	s12 = rotate64(s21, (uint64_t) 6); \
	s21 = rotate64(s02, (uint64_t) 3); \
	s02 = t0; \
}

#define KHI(s00, s01, s02, s03, s04, \
            s10, s11, s12, s13, s14, \
            s20, s21, s22, s23, s24, \
            s30, s31, s32, s33, s34, \
            s40, s41, s42, s43, s44) \
{ \
    t0 = s00 ^ (~s10 &  s20); \
    t1 = s10 ^ (~s20 &  s30); \
    t2 = s20 ^ (~s30 &  s40); \
    t3 = s30 ^ (~s40 &  s00); \
    t4 = s40 ^ (~s00 &  s10); \
    s00 = t0; s10 = t1; s20 = t2; s30 = t3; s40 = t4; \
    \
    t0 = s01 ^ (~s11 &  s21); \
    t1 = s11 ^ (~s21 &  s31); \
    t2 = s21 ^ (~s31 &  s41); \
    t3 = s31 ^ (~s41 &  s01); \
    t4 = s41 ^ (~s01 &  s11); \
    s01 = t0; s11 = t1; s21 = t2; s31 = t3; s41 = t4; \
    \
    t0 = s02 ^ (~s12 &  s22); \
    t1 = s12 ^ (~s22 &  s32); \
    t2 = s22 ^ (~s32 &  s42); \
    t3 = s32 ^ (~s42 &  s02); \
    t4 = s42 ^ (~s02 &  s12); \
    s02 = t0; s12 = t1; s22 = t2; s32 = t3; s42 = t4; \
    \
    t0 = s03 ^ (~s13 &  s23); \
    t1 = s13 ^ (~s23 &  s33); \
    t2 = s23 ^ (~s33 &  s43); \
    t3 = s33 ^ (~s43 &  s03); \
    t4 = s43 ^ (~s03 &  s13); \
    s03 = t0; s13 = t1; s23 = t2; s33 = t3; s43 = t4; \
    \
    t0 = s04 ^ (~s14 &  s24); \
    t1 = s14 ^ (~s24 &  s34); \
    t2 = s24 ^ (~s34 &  s44); \
    t3 = s34 ^ (~s44 &  s04); \
    t4 = s44 ^ (~s04 &  s14); \
    s04 = t0; s14 = t1; s24 = t2; s34 = t3; s44 = t4; \
}

#define IOTA(s00, r) { s00 ^= r; }

__constant__ uint64_t keccakf_rndc[24] = {
	0x0000000000000001, 0x0000000000008082, 0x800000000000808a,
	0x8000000080008000, 0x000000000000808b, 0x0000000080000001,
	0x8000000080008081, 0x8000000000008009, 0x000000000000008a,
	0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
	0x000000008000808b, 0x800000000000008b, 0x8000000000008089,
	0x8000000000008003, 0x8000000000008002, 0x8000000000000080,
	0x000000000000800a, 0x800000008000000a, 0x8000000080008081,
	0x8000000000008080, 0x0000000080000001, 0x8000000080008008
};

// Barely a bottleneck. No need to tinker more.
__device__
void sha3_keccakf(ethhash* const h)
{
	uint64_t* const st = (uint64_t* const)&h->q;
	h->d[33] ^= 0x80000000;
	uint64_t t0, t1, t2, t3, t4;

	// Unrolling and removing PI stage gave negligable performance on GTX 1070.
	for (int i = 0; i < 24; ++i) {
		THETA(st[0], st[5], st[10], st[15], st[20], st[1], st[6], st[11], st[16], st[21], st[2], st[7], st[12], st[17], st[22], st[3], st[8], st[13], st[18], st[23], st[4], st[9], st[14], st[19], st[24]);
		RHOPI(st[0], st[5], st[10], st[15], st[20], st[1], st[6], st[11], st[16], st[21], st[2], st[7], st[12], st[17], st[22], st[3], st[8], st[13], st[18], st[23], st[4], st[9], st[14], st[19], st[24]);
		KHI(st[0], st[5], st[10], st[15], st[20], st[1], st[6], st[11], st[16], st[21], st[2], st[7], st[12], st[17], st[22], st[3], st[8], st[13], st[18], st[23], st[4], st[9], st[14], st[19], st[24]);
		IOTA(st[0], keccakf_rndc[i]);
	}
}

/* ------------------------------------------------------------------------ */
/* Multiprecision functions                                                 */
/* ------------------------------------------------------------------------ */
#define MP_WORDS 8
#define MP_BITS 32
#define bswap32(n) (rotate32(n & 0x00FF00FF, 24U)|(rotate32(n, 8U) & 0x00FF00FF))

// mod              = 0xfffffffffffffffffffffffffffffffffffffffffffffffffffffffefffffc2f
__constant__ const mp_number mod = { {0xfffffc2f, 0xfffffffe, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff} };

// Multiprecision subtraction. Underflow signalled via return value.
__device__
uint32_t mp_sub(mp_number* const r, const mp_number* const a, const mp_number* const b) {
	uint32_t t, c = 0;

	for (uint32_t i = 0; i < MP_WORDS; ++i) {
		t = a->d[i] - b->d[i] - c;
		c = t > a->d[i] ? 1 : (t == a->d[i] ? c : 0);

		r->d[i] = t;
	}

	return c;
}

// Multiprecision subtraction of the modulus saved in mod. Underflow signalled via return value.
__device__
uint32_t mp_sub_mod(mp_number* const r) {
	mp_number mod = { {0xfffffc2f, 0xfffffffe, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff} };

	uint32_t t, c = 0;

	for (uint32_t i = 0; i < MP_WORDS; ++i) {
		t = r->d[i] - mod.d[i] - c;
		c = t > r->d[i] ? 1 : (t == r->d[i] ? c : 0);

		r->d[i] = t;
	}

	return c;
}

// Multiprecision subtraction modulo M, M = mod.
// This function is often also used for additions by subtracting a negative number. I've chosen
// to do this because:
//   1. It's easier to re-use an already existing function
//   2. A modular addition would have more overhead since it has to determine if the result of
//      the addition (r) is in the gap M <= r < 2^256. This overhead doesn't exist in a
//      subtraction. We immediately know at the end of a subtraction if we had underflow
//      or not by inspecting the carry value. M refers to the modulus saved in variable mod.
__device__
void mp_mod_sub(mp_number* const r, const mp_number* const a, const mp_number* const b) {
	uint32_t i, t, c = 0;

	for (i = 0; i < MP_WORDS; ++i) {
		t = a->d[i] - b->d[i] - c;
		c = t < a->d[i] ? 0 : (t == a->d[i] ? c : 1);

		r->d[i] = t;
	}

	if (c) {
		c = 0;
		for (i = 0; i < MP_WORDS; ++i) {
			r->d[i] += mod.d[i] + c;
			c = r->d[i] < mod.d[i] ? 1 : (r->d[i] == mod.d[i] ? c : 0);
		}
	}
}

// Multiprecision subtraction modulo M from a constant number.
// I made this in the belief that using constant address space instead of private address space for any
// constant numbers would lead to increase in performance. Judges are still out on this one.
__device__
void mp_mod_sub_const(mp_number* const r, const mp_number* const a, const mp_number* const b) {
	uint32_t i, t, c = 0;

	for (i = 0; i < MP_WORDS; ++i) {
		t = a->d[i] - b->d[i] - c;
		c = t < a->d[i] ? 0 : (t == a->d[i] ? c : 1);

		r->d[i] = t;
	}

	if (c) {
		c = 0;
		for (i = 0; i < MP_WORDS; ++i) {
			r->d[i] += mod.d[i] + c;
			c = r->d[i] < mod.d[i] ? 1 : (r->d[i] == mod.d[i] ? c : 0);
		}
	}
}

// Multiprecision subtraction modulo M of G_x from a number.
// Specialization of mp_mod_sub in hope of performance gain.
__device__
void mp_mod_sub_gx(mp_number* const r, const mp_number* const a) {
	uint32_t i, t, c = 0;

	t = a->d[0] - 0x16f81798; c = t < a->d[0] ? 0 : (t == a->d[0] ? c : 1); r->d[0] = t;
	t = a->d[1] - 0x59f2815b - c; c = t < a->d[1] ? 0 : (t == a->d[1] ? c : 1); r->d[1] = t;
	t = a->d[2] - 0x2dce28d9 - c; c = t < a->d[2] ? 0 : (t == a->d[2] ? c : 1); r->d[2] = t;
	t = a->d[3] - 0x029bfcdb - c; c = t < a->d[3] ? 0 : (t == a->d[3] ? c : 1); r->d[3] = t;
	t = a->d[4] - 0xce870b07 - c; c = t < a->d[4] ? 0 : (t == a->d[4] ? c : 1); r->d[4] = t;
	t = a->d[5] - 0x55a06295 - c; c = t < a->d[5] ? 0 : (t == a->d[5] ? c : 1); r->d[5] = t;
	t = a->d[6] - 0xf9dcbbac - c; c = t < a->d[6] ? 0 : (t == a->d[6] ? c : 1); r->d[6] = t;
	t = a->d[7] - 0x79be667e - c; c = t < a->d[7] ? 0 : (t == a->d[7] ? c : 1); r->d[7] = t;

	if (c) {
		c = 0;
		for (i = 0; i < MP_WORDS; ++i) {
			r->d[i] += mod.d[i] + c;
			c = r->d[i] < mod.d[i] ? 1 : (r->d[i] == mod.d[i] ? c : 0);
		}
	}
}

//__device__
//void mp_mod_sub_gx(mp_number* const r, const mp_number* const a) {
//	uint32_t i, t, c = 0;
//
//	t = a->d[0] - 0xe907e497; c = t < a->d[0] ? 0 : (t == a->d[0] ? c : 1); r->d[0] = t;
//	t = a->d[1] - 0xa60d7ea3 - c; c = t < a->d[1] ? 0 : (t == a->d[1] ? c : 1); r->d[1] = t;
//	t = a->d[2] - 0xd231d726 - c; c = t < a->d[2] ? 0 : (t == a->d[2] ? c : 1); r->d[2] = t;
//	t = a->d[3] - 0xfd640324 - c; c = t < a->d[3] ? 0 : (t == a->d[3] ? c : 1); r->d[3] = t;
//	t = a->d[4] - 0x3178f4f8 - c; c = t < a->d[4] ? 0 : (t == a->d[4] ? c : 1); r->d[4] = t;
//	t = a->d[5] - 0xaa5f9d6a - c; c = t < a->d[5] ? 0 : (t == a->d[5] ? c : 1); r->d[5] = t;
//	t = a->d[6] - 0x06234453 - c; c = t < a->d[6] ? 0 : (t == a->d[6] ? c : 1); r->d[6] = t;
//	t = a->d[7] - 0x86419981 - c; c = t < a->d[7] ? 0 : (t == a->d[7] ? c : 1); r->d[7] = t;
//
//	if (c) {
//		c = 0;
//		for (i = 0; i < MP_WORDS; ++i) {
//			r->d[i] += mod.d[i] + c;
//			c = r->d[i] < mod.d[i] ? 1 : (r->d[i] == mod.d[i] ? c : 0);
//		}
//	}
//}

// Multiprecision subtraction modulo M of G_y from a number.
// Specialization of mp_mod_sub in hope of performance gain.
//__device__
//void mp_mod_sub_gy(mp_number* const r, const mp_number* const a) {
//	uint32_t i, t, c = 0;
//
//	t = a->d[0] - 0xfb10d4b8; c = t < a->d[0] ? 0 : (t == a->d[0] ? c : 1); r->d[0] = t;
//	t = a->d[1] - 0x9c47d08f - c; c = t < a->d[1] ? 0 : (t == a->d[1] ? c : 1); r->d[1] = t;
//	t = a->d[2] - 0xa6855419 - c; c = t < a->d[2] ? 0 : (t == a->d[2] ? c : 1); r->d[2] = t;
//	t = a->d[3] - 0xfd17b448 - c; c = t < a->d[3] ? 0 : (t == a->d[3] ? c : 1); r->d[3] = t;
//	t = a->d[4] - 0x0e1108a8 - c; c = t < a->d[4] ? 0 : (t == a->d[4] ? c : 1); r->d[4] = t;
//	t = a->d[5] - 0x5da4fbfc - c; c = t < a->d[5] ? 0 : (t == a->d[5] ? c : 1); r->d[5] = t;
//	t = a->d[6] - 0x26a3c465 - c; c = t < a->d[6] ? 0 : (t == a->d[6] ? c : 1); r->d[6] = t;
//	t = a->d[7] - 0x483ada77 - c; c = t < a->d[7] ? 0 : (t == a->d[7] ? c : 1); r->d[7] = t;
//
//	if (c) {
//		c = 0;
//		for (i = 0; i < MP_WORDS; ++i) {
//			r->d[i] += mod.d[i] + c;
//			c = r->d[i] < mod.d[i] ? 1 : (r->d[i] == mod.d[i] ? c : 0);
//		}
//	}
//}

__device__
void mp_mod_sub_gy(mp_number* const r, const mp_number* const a) {
	uint32_t i, t, c = 0;

	t = a->d[0] - 0x04ef2777; c = t < a->d[0] ? 0 : (t == a->d[0] ? c : 1); r->d[0] = t;
	t = a->d[1] - 0x63b82f6f - c; c = t < a->d[1] ? 0 : (t == a->d[1] ? c : 1); r->d[1] = t;
	t = a->d[2] - 0x597aabe6 - c; c = t < a->d[2] ? 0 : (t == a->d[2] ? c : 1); r->d[2] = t;
	t = a->d[3] - 0x02e84bb7 - c; c = t < a->d[3] ? 0 : (t == a->d[3] ? c : 1); r->d[3] = t;
	t = a->d[4] - 0xf1eef757 - c; c = t < a->d[4] ? 0 : (t == a->d[4] ? c : 1); r->d[4] = t;
	t = a->d[5] - 0xa25b0403 - c; c = t < a->d[5] ? 0 : (t == a->d[5] ? c : 1); r->d[5] = t;
	t = a->d[6] - 0xd95c3b9a - c; c = t < a->d[6] ? 0 : (t == a->d[6] ? c : 1); r->d[6] = t;
	t = a->d[7] - 0xb7c52588 - c; c = t < a->d[7] ? 0 : (t == a->d[7] ? c : 1); r->d[7] = t;

	if (c) {
		c = 0;
		for (i = 0; i < MP_WORDS; ++i) {
			r->d[i] += mod.d[i] + c;
			c = r->d[i] < mod.d[i] ? 1 : (r->d[i] == mod.d[i] ? c : 0);
		}
	}
}




// Multiprecision addition. Overflow signalled via return value.
__device__
uint32_t mp_add(mp_number* const r, const mp_number* const a) {
	uint32_t c = 0;

	for (uint32_t i = 0; i < MP_WORDS; ++i) {
		r->d[i] += a->d[i] + c;
		c = r->d[i] < a->d[i] ? 1 : (r->d[i] == a->d[i] ? c : 0);
	}

	return c;
}

// Multiprecision addition. Overflow signalled via return value.
__device__
uint32_t mp_add_size(mp_number* const r, const uint32_t a) {
	uint32_t c = 0;
	r->d[0] += a;
	c = r->d[0] < a ? 1 : (r->d[0] == a ? c : 0);
	for (uint32_t i = 1; i < MP_WORDS; ++i) {
		r->d[i] += c;
		c = r->d[i] == 0 ? 1 : 0;
		if (c == 0) break;
	}

	return c;
}


// Multiprecision addition of the modulus saved in mod. Overflow signalled via return value.
__device__
uint32_t mp_add_mod(mp_number* const r) {
	uint32_t c = 0;

	for (uint32_t i = 0; i < MP_WORDS; ++i) {
		r->d[i] += mod.d[i] + c;
		c = r->d[i] < mod.d[i] ? 1 : (r->d[i] == mod.d[i] ? c : 0);
	}

	return c;
}


__device__
uint32_t mp_mod_add(mp_number* const r, const mp_number* const a) {
	uint32_t c = 0;
	c = mp_add(r, a);
	if (c) {
		mp_sub_mod(r);
	}
	return c;
}

// Multiprecision addition of two numbers with one extra word each. Overflow signalled via return value.
__device__
uint32_t mp_add_more(mp_number* const r, uint32_t* const extraR, const mp_number* const a, const uint32_t* const extraA) {
	const uint32_t c = mp_add(r, a);
	*extraR += *extraA + c;
	return *extraR < *extraA ? 1 : (*extraR == *extraA ? c : 0);
}

// Multiprecision greater than or equal (>=) operator
__device__
uint32_t mp_gte(const mp_number* const a, const mp_number* const b) {
	uint32_t l = 0, g = 0;

	for (uint32_t i = 0; i < MP_WORDS; ++i) {
		if (a->d[i] < b->d[i]) l |= (1 << i);
		if (a->d[i] > b->d[i]) g |= (1 << i);
	}

	return g >= l;
}

// Bit shifts a number with an extra word to the right one step
__device__
void mp_shr_extra(mp_number* const r, uint32_t* const e) {
	r->d[0] = (r->d[1] << 31) | (r->d[0] >> 1);
	r->d[1] = (r->d[2] << 31) | (r->d[1] >> 1);
	r->d[2] = (r->d[3] << 31) | (r->d[2] >> 1);
	r->d[3] = (r->d[4] << 31) | (r->d[3] >> 1);
	r->d[4] = (r->d[5] << 31) | (r->d[4] >> 1);
	r->d[5] = (r->d[6] << 31) | (r->d[5] >> 1);
	r->d[6] = (r->d[7] << 31) | (r->d[6] >> 1);
	r->d[7] = (*e << 31) | (r->d[7] >> 1);
	*e >>= 1;
}

// Bit shifts a number to the right one step
__device__
void mp_shr(mp_number* const r) {
	r->d[0] = (r->d[1] << 31) | (r->d[0] >> 1);
	r->d[1] = (r->d[2] << 31) | (r->d[1] >> 1);
	r->d[2] = (r->d[3] << 31) | (r->d[2] >> 1);
	r->d[3] = (r->d[4] << 31) | (r->d[3] >> 1);
	r->d[4] = (r->d[5] << 31) | (r->d[4] >> 1);
	r->d[5] = (r->d[6] << 31) | (r->d[5] >> 1);
	r->d[6] = (r->d[7] << 31) | (r->d[6] >> 1);
	r->d[7] >>= 1;
}

// Multiplies a number with a word and adds it to an existing number with an extra word, overflow of the extra word is signalled in return value
// This is a special function only used for modular multiplication
__device__
uint32_t mp_mul_word_add_extra(mp_number* const r, const mp_number* const a, const uint32_t w, uint32_t* const extra) {
	uint32_t cM = 0; // Carry for multiplication
	uint32_t cA = 0; // Carry for addition
	uint32_t tM = 0; // Temporary storage for multiplication

	for (uint32_t i = 0; i < MP_WORDS; ++i) {
		tM = (a->d[i] * w + cM);
		cM = mul_hi(a->d[i], w) + (tM < cM);

		r->d[i] += tM + cA;
		cA = r->d[i] < tM ? 1 : (r->d[i] == tM ? cA : 0);
	}

	*extra += cM + cA;
	return *extra < cM ? 1 : (*extra == cM ? cA : 0);
}

// Multiplies a number with a word, potentially adds modhigher to it, and then subtracts it from en existing number, no extra words, no overflow
// This is a special function only used for modular multiplication
__device__
void mp_mul_mod_word_sub(mp_number* const r, const uint32_t w, const bool withModHigher) {
	// Having these numbers declared here instead of using the global values in __constant__ address space seems to lead
	// to better optimizations by the compiler on my GTX 1070.
	mp_number mod = { { 0xfffffc2f, 0xfffffffe, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff} };
	mp_number modhigher = { {0x00000000, 0xfffffc2f, 0xfffffffe, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff} };

	uint32_t cM = 0; // Carry for multiplication
	uint32_t cS = 0; // Carry for subtraction
	uint32_t tS = 0; // Temporary storage for subtraction
	uint32_t tM = 0; // Temporary storage for multiplication
	uint32_t cA = 0; // Carry for addition of modhigher

	for (uint32_t i = 0; i < MP_WORDS; ++i) {
		tM = (mod.d[i] * w + cM);
		cM = mul_hi(mod.d[i], w) + (tM < cM);

		tM += (withModHigher ? modhigher.d[i] : 0) + cA;
		cA = tM < (withModHigher ? modhigher.d[i] : 0) ? 1 : (tM == (withModHigher ? modhigher.d[i] : 0) ? cA : 0);

		tS = r->d[i] - tM - cS;
		cS = tS > r->d[i] ? 1 : (tS == r->d[i] ? cS : 0);

		r->d[i] = tS;
	}
}

// Modular multiplication. Based on Algorithm 3 (and a series of hunches) from this article:
// https://www.esat.kuleuven.be/cosic/publications/article-1191.pdf
// When I first implemented it I never encountered a situation where the additional end steps
// of adding or subtracting the modulo was necessary. Maybe it's not for the particular modulo
// used in secp256k1, maybe the overflow bit can be skipped in to avoid 8 subtractions and
// trade it for the final steps? Maybe the final steps are necessary but seldom needed?
// I have no idea, for the time being I'll leave it like this, also see the comments at the
// beginning of this document under the title "Cutting corners".
__device__
void mp_mod_mul(mp_number* const r, const mp_number* const X, const mp_number* const Y) {
	mp_number Z = { {0} };
	uint32_t extraWord;

	for (int i = MP_WORDS - 1; i >= 0; --i) {
		// Z = Z * 2^32
		extraWord = Z.d[7]; Z.d[7] = Z.d[6]; Z.d[6] = Z.d[5]; Z.d[5] = Z.d[4]; Z.d[4] = Z.d[3]; Z.d[3] = Z.d[2]; Z.d[2] = Z.d[1]; Z.d[1] = Z.d[0]; Z.d[0] = 0;

		// Z = Z + X * Y_i
		bool overflow = mp_mul_word_add_extra(&Z, X, Y->d[i], &extraWord);

		// Z = Z - qM
		mp_mul_mod_word_sub(&Z, extraWord, overflow);
	}

	*r = Z;
}


// Modular inversion of a number. 
__device__
void mp_mod_inverse(mp_number* const r) {
	mp_number A = { { 1 } };
	mp_number C = { { 0 } };
	mp_number v = mod;

	uint32_t extraA = 0;
	uint32_t extraC = 0;

	while (r->d[0] || r->d[1] || r->d[2] || r->d[3] || r->d[4] || r->d[5] || r->d[6] || r->d[7]) {
		while (!(r->d[0] & 1)) {
			mp_shr(r);
			if (A.d[0] & 1) {
				extraA += mp_add_mod(&A);
			}

			mp_shr_extra(&A, &extraA);
		}

		while (!(v.d[0] & 1)) {
			mp_shr(&v);
			if (C.d[0] & 1) {
				extraC += mp_add_mod(&C);
			}

			mp_shr_extra(&C, &extraC);
		}

		if (mp_gte(r, &v)) {
			mp_sub(r, r, &v);
			mp_add_more(&A, &extraA, &C, &extraC);
		}
		else {
			mp_sub(&v, &v, r);
			mp_add_more(&C, &extraC, &A, &extraA);
		}
	}

	while (extraC) {
		extraC -= mp_sub_mod(&C);
	}

	v = mod;
	mp_sub(r, &v, &C);
}

__device__
void mp_mod_div(mp_number* const r, const mp_number* const X, const mp_number* const Y) {
	mp_number inverse;
	inverse.d[0] = Y->d[0];
	inverse.d[1] = Y->d[1];
	inverse.d[2] = Y->d[2];
	inverse.d[3] = Y->d[3];
	inverse.d[4] = Y->d[4];
	inverse.d[5] = Y->d[5];
	inverse.d[6] = Y->d[6];
	inverse.d[7] = Y->d[7];
	mp_mod_inverse(&inverse);
	mp_mod_mul(r, X, &inverse);
}
/* ------------------------------------------------------------------------ */
/* Elliptic point and addition (with caveats).                              */
/* ------------------------------------------------------------------------ */

// Elliptical point addition
// Does not handle points sharing X coordinate, this is a deliberate design choice.
// For more information on this choice see the beginning of this file.
__device__
void point_add(point* const r, point* const p, point* const o) {
	mp_number tmp;
	mp_number newX;
	mp_number newY;

	mp_mod_sub(&tmp, &o->x, &p->x);

	mp_mod_inverse(&tmp);

	mp_mod_sub(&newX, &o->y, &p->y);
	mp_mod_mul(&tmp, &tmp, &newX);

	mp_mod_mul(&newX, &tmp, &tmp);
	mp_mod_sub(&newX, &newX, &p->x);
	mp_mod_sub(&newX, &newX, &o->x);

	mp_mod_sub(&newY, &p->x, &newX);
	mp_mod_mul(&newY, &newY, &tmp);
	mp_mod_sub(&newY, &newY, &p->y);

	r->x = newX;
	r->y = newY;
}



__device__
void printMpNumber(mp_number* x) {
	printf("%.8x%.8x%.8x%.8x%.8x%.8x%.8x%.8x\n",
		x->d[7],
		x->d[6],
		x->d[5],
		x->d[4],
		x->d[3],
		x->d[2],
		x->d[1],
		x->d[0]);
}

__device__
void printPoint(point* p) {
	printf("X: %.8x%.8x%.8x%.8x%.8x%.8x%.8x%.8x\nY: %.8x%.8x%.8x%.8x%.8x%.8x%.8x%.8x\n",
		p->x.d[7],
		p->x.d[6],
		p->x.d[5],
		p->x.d[4],
		p->x.d[3],
		p->x.d[2],
		p->x.d[1],
		p->x.d[0],
		p->y.d[7],
		p->y.d[6],
		p->y.d[5],
		p->y.d[4],
		p->y.d[3],
		p->y.d[2],
		p->y.d[1],
		p->y.d[0]);
}


__device__
void mul_G(const point* const precomp, point* const p, const size_t precompOffset, const uint64_t k) {
	point o;
	bool bIsFirst = true;
	for (uint8_t i = 0; i < 8; ++i) {
		const uint8_t shift = i * 8;
		const uint8_t byte = (k >> shift) & 0xFF;

		if (byte) {
			o = precomp[precompOffset + i * 255 + byte - 1];
			if (bIsFirst) {
				*p = o;
				bIsFirst = false;
			}
			else {
				point_add(p, p, &o);
			}
		}
	}
}

__global__ void dev_crack_init(const point* const precomp, point* const extensionPublicKey, point* publicKey) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	point pubKey; //public key = publicKey - G*2^192*id
	point g2192; //G*2^192
	for (int i = 0; i < 8; i++)
	{
		pubKey.x.d[i] = publicKey->x.d[i];
		pubKey.y.d[i] = publicKey->y.d[i];
	}
	if (idx != 0) {
		uint32_t num = idx;
		mul_G(precomp, &g2192, 8 * 255 * 3, num); //calculate G*2^192*id
		//-G*2^192*id
		mp_number zero = { {0} };
		mp_mod_sub(&g2192.y, &zero, &g2192.y); //calculate negative y coordinate
		point_add(&pubKey, &pubKey, &g2192); //publicKey + (-G*2^192*id)
	}
	for (int ii = 0; ii < 8; ii++)
	{
		extensionPublicKey[idx].x.d[ii] = pubKey.x.d[ii];
		extensionPublicKey[idx].y.d[ii] = pubKey.y.d[ii];
	}
}





/* Four of six logical functions used in SHA-384 and SHA-512: */
#define REVERSE32_FOR_HASH(w,x)	{ \
	uint32_t tmp = (w); \
	tmp = (tmp >> 16) | (tmp << 16); \
	(x) = ((tmp & 0xff00ff00UL) >> 8) | ((tmp & 0x00ff00ffUL) << 8); \
}
#define REVERSE64_FOR_HASH(w,x)	{ \
	uint64_t tmp = (w); \
	tmp = (tmp >> 32) | (tmp << 32); \
	tmp = ((tmp & 0xff00ff00ff00ff00UL) >> 8) | \
	      ((tmp & 0x00ff00ff00ff00ffUL) << 8); \
	(x) = ((tmp & 0xffff0000ffff0000UL) >> 16) | \
	      ((tmp & 0x0000ffff0000ffffUL) << 16); \
}

int comp_key_test(const point* const key, const uint32_t bytes_y_from_table) {
	const uint32_t bytes_y = key->x.d[7];
	//const uint64_t bytes_y = 1;
	if (bytes_y < bytes_y_from_table)
	{
		return -1;
	}
	else if (bytes_y > bytes_y_from_table)
	{
		return 1;
	}
	return 0;
}


__device__
int comp_key(const point* const key, const uint64_t bytes_y_from_table) {
	uint64_t bytes_y = *(const uint64_t*)&key->x.d[6];
	uint64_t bytes_x_from_table = bytes_y_from_table;

	if (bytes_y < bytes_x_from_table)
	{
		return -1;
	}
	else if (bytes_y > bytes_x_from_table)
	{
		return 1;
	}
	return 0;
}


__constant__ const mp_number tripleNegativeGx = { {0xbb17b196, 0xf2287bec, 0x76958573, 0xf82c096e, 0x946adeea, 0xff1ed83e, 0x1269ccfa, 0x92c4cc83 } };
__constant__ mp_number negativeDoubleGy = { { 0xf621a970, 0x388fa11f, 0x4d0aa833, 0xfa2f6891, 0x1c221151, 0xbb49f7f8, 0x4d4788ca, 0x9075b4ee } };
__constant__ const mp_number negativeGy = { {0xfb10d4b8, 0x9c47d08f, 0xa6855419, 0xfd17b448, 0x0e1108a8, 0x5da4fbfc, 0x26a3c465, 0x483ada77 } };
__constant__ mp_number negativeGx = { {0xe907e497, 0xa60d7ea3, 0xd231d726, 0xfd640324, 0x3178f4f8, 0xaa5f9d6a, 0x06234453, 0x86419981 } };
__constant__ point negativeG = { { 0x16f81798, 0x59f2815b, 0x2dce28d9, 0x029bfcdb, 0xce870b07, 0x55a06295, 0xf9dcbbac, 0x79be667e },{0x04ef2777, 0x63b82f6f, 0x597aabe6, 0x02e84bb7, 0xf1eef757, 0xa25b0403, 0xd95c3b9a, 0xb7c52588 } };



__device__
int find_in_table_8_bytes(const point* const key, const tableStruct* table, uint64_t* line_in_table_find) {
	int find = 0;
	bool search_state = true;
	uint64_t line_cnt = table->size / 8;
	uint64_t interval = line_cnt / 3;

	uint64_t num_line_next = 0;
	uint64_t num_line_last = 0;

	while (num_line_next < line_cnt) {
		num_line_last = num_line_next;
		uint64_t bytes_from_table;


		if (interval == 0) {
			search_state = false;
		}
		if (search_state) {
			num_line_next += interval;

			if (num_line_next >= line_cnt) {
				num_line_next = num_line_last;
				interval = interval / 2;
				continue;
			}
			bytes_from_table = table->table[num_line_next];
			*line_in_table_find = num_line_next;
		}
		else {
			bytes_from_table = table->table[num_line_next];
			*line_in_table_find = num_line_next;
			num_line_next += 1;
		}
		int cmp = comp_key(key, bytes_from_table);
		//REVERSE32_FOR_HASH(bytes_from_table, bytes_from_table);
		//int cmp = comp_key_test(key, bytes_from_table);
		if (search_state) {
			if (cmp < 0) {
				if (interval < 20) {
					search_state = false;
				}
				else
				{
					interval = interval / 2;
				}
				num_line_next = num_line_last;
				continue;
			}
			else if (cmp == 0) {
				search_state = false;
			}
			else {
				continue;
			}
		}


		if (cmp <= 0) {
			if (cmp == 0)
				find = 1;
			break;
		}
	}

	if (find == 1) {
		return 1;
	}
	return 0;
}

__global__ void dev_crack_init(const point* const precomp, mp_number* const pDeltaX, mp_number* const pPrevLambda, point* publicKey) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	mp_number tmp1, tmp2;
	//point tmp3;

	point pubKey; //public key = publicKey - G*2^192*id
	point g2192; //G*2^192
	for (int i = 0; i < 8; i++)
	{
		pubKey.x.d[i] = publicKey->x.d[i];
		pubKey.y.d[i] = publicKey->y.d[i];
	}
	if (idx != 0) {
		uint32_t num = idx;
		mul_G(precomp, &g2192, 8 * 255 * 3, num); //calculate G*2^192*id
		//-G*2^192*id
		mp_number zero = { {0} };
		mp_mod_sub(&g2192.y, &zero, &g2192.y); //calculate negative y coordinate
		point_add(&pubKey, &pubKey, &g2192); //publicKey + (-G*2^192*id)
	}

	// Calculate current lambda in this point
	mp_mod_sub_gx(&tmp1, &pubKey.x);
	mp_mod_inverse(&tmp1);

	mp_mod_sub_gy(&tmp2, &pubKey.y);
	mp_mod_mul(&tmp1, &tmp1, &tmp2);

	// Jump to next point (precomp[0] is the generator point G)
	//tmp3 = precomp[0];
	point_add(&pubKey, &negativeG, &pubKey);
	// pDeltaX should contain the delta (x - G_x)
	mp_mod_sub_gx(&pubKey.x, &pubKey.x);

	pDeltaX[idx] = pubKey.x;
	pPrevLambda[idx] = tmp1;

}

__global__ void profanity_inverse(const mp_number* const pDeltaX, mp_number* const pInverse) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const size_t id = idx * PROFANITY_INVERSE_SIZE;

	mp_number copy1, copy2;
	mp_number buffer[PROFANITY_INVERSE_SIZE];
	mp_number buffer2[PROFANITY_INVERSE_SIZE];

	// We initialize buffer and buffer2 such that:
	// buffer[i] = pDeltaX[id] * pDeltaX[id + 1] * pDeltaX[id + 2] * ... * pDeltaX[id + i]
	// buffer2[i] = pDeltaX[id + i]
	buffer[0] = pDeltaX[id];
	for (uint32_t i = 1; i < PROFANITY_INVERSE_SIZE; ++i) {
		buffer2[i] = pDeltaX[id + i];
		mp_mod_mul(&buffer[i], &buffer2[i], &buffer[i - 1]);
	}

	// Take the inverse of all x-values combined
	copy1 = buffer[PROFANITY_INVERSE_SIZE - 1];
	mp_mod_inverse(&copy1);

	// We multiply in -2G_y together with the inverse so that we have:
	//            - 2 * G_y
	//  ----------------------------
	//  x_0 * x_1 * x_2 * x_3 * ...
	mp_mod_mul(&copy1, &copy1, &negativeDoubleGy);

	// Multiply out each individual inverse using the buffers
	for (uint32_t i = PROFANITY_INVERSE_SIZE - 1; i > 0; --i) {
		mp_mod_mul(&copy2, &copy1, &buffer[i - 1]);
		mp_mod_mul(&copy1, &copy1, &buffer2[i]);
		pInverse[id + i] = copy2;
	}

	pInverse[id] = copy1;
}

__global__ void dev_crack_search_only_gpu(mp_number* const pDeltaX, mp_number* const pInverse, mp_number* const pPrevLambda, const tableStruct* __restrict__ tables, resultFound* __restrict__ m_memResult) {
	const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;;

	mp_number dX = pDeltaX[idx];
	mp_number tmp = pInverse[idx];
	mp_number lambda = pPrevLambda[idx];

	// λ' = - (2G_y) / d' - λ <=> lambda := pInversedNegativeDoubleGy[id] - pPrevLambda[id]
	mp_mod_sub(&lambda, &tmp, &lambda);

	// λ² = λ * λ <=> tmp := lambda * lambda = λ²
	mp_mod_mul(&tmp, &lambda, &lambda);

	// d' = λ² - d - 3g = (-3g) - (d - λ²) <=> x := tripleNegativeGx - (x - tmp)
	mp_mod_sub(&dX, &dX, &tmp);
	mp_mod_sub_const(&dX, &tripleNegativeGx, &dX);

	pDeltaX[idx] = dX;
	pPrevLambda[idx] = lambda;

	// Calculate y from dX and lambda
	// y' = (-G_Y) - λ * d' <=> p.y := negativeGy - (p.y * p.x)
	mp_mod_mul(&tmp, &lambda, &dX);
	mp_mod_sub_const(&tmp, &negativeGy, &tmp);

	// Restore X coordinate from delta value
	mp_mod_sub(&dX, &dX, &negativeGx);
	point pubKey;
	pubKey.x.d[0] = dX.d[0];
	pubKey.x.d[1] = dX.d[1];
	pubKey.x.d[2] = dX.d[2];
	pubKey.x.d[3] = dX.d[3];
	pubKey.x.d[4] = dX.d[4];
	pubKey.x.d[5] = dX.d[5];
	pubKey.x.d[6] = dX.d[6];
	pubKey.x.d[7] = dX.d[7];
	pubKey.y.d[0] = tmp.d[0];
	pubKey.y.d[1] = tmp.d[1];
	pubKey.y.d[2] = tmp.d[2];
	pubKey.y.d[3] = tmp.d[3];
	pubKey.y.d[4] = tmp.d[4];
	pubKey.y.d[5] = tmp.d[5];
	pubKey.y.d[6] = tmp.d[6];
	pubKey.y.d[7] = tmp.d[7];



	uint8_t num_table = *(uint8_t*)((uint8_t*)&pubKey.x.d[7] + 3);
	int ret = 0;
	uint64_t line_in_table_find;
	if (tables[num_table].size != 0)
	{
		ret = find_in_table_8_bytes(&pubKey, &tables[num_table], &line_in_table_find);
	}
	if (ret == 1)
	{
		m_memResult->score++;
		if (m_memResult->score >= NUM_RESULT_FOUND_KEYS)
		{
			m_memResult->score = 0;
		}
		uint32_t score = m_memResult->score;
		for (int ii = 0; ii < 8; ii++)
		{
			m_memResult->pub_key[score].x.d[ii] = pubKey.x.d[ii];
			m_memResult->pub_key[score].y.d[ii] = pubKey.y.d[ii];
		}
		m_memResult->line_in_tables[score] = line_in_table_find;
		m_memResult->id[score] = idx;
	}
}
__global__ void dev_crack_search_only_cpu(mp_number* const pDeltaX, mp_number* const pInverse, mp_number* const pPrevLambda, point* __restrict__ m_memResult) {
	const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;;

	mp_number dX = pDeltaX[idx];
	mp_number tmp = pInverse[idx];
	mp_number lambda = pPrevLambda[idx];

	// λ' = - (2G_y) / d' - λ <=> lambda := pInversedNegativeDoubleGy[id] - pPrevLambda[id]
	mp_mod_sub(&lambda, &tmp, &lambda);

	// λ² = λ * λ <=> tmp := lambda * lambda = λ²
	mp_mod_mul(&tmp, &lambda, &lambda);

	// d' = λ² - d - 3g = (-3g) - (d - λ²) <=> x := tripleNegativeGx - (x - tmp)
	mp_mod_sub(&dX, &dX, &tmp);
	mp_mod_sub_const(&dX, &tripleNegativeGx, &dX);

	pDeltaX[idx] = dX;
	pPrevLambda[idx] = lambda;

	// Calculate y from dX and lambda
	// y' = (-G_Y) - λ * d' <=> p.y := negativeGy - (p.y * p.x)
	mp_mod_mul(&tmp, &lambda, &dX);
	mp_mod_sub_const(&tmp, &negativeGy, &tmp);

	// Restore X coordinate from delta value
	mp_mod_sub(&dX, &dX, &negativeGx);
	m_memResult[idx].x.d[0] = dX.d[0];
	m_memResult[idx].x.d[1] = dX.d[1];
	m_memResult[idx].x.d[2] = dX.d[2];
	m_memResult[idx].x.d[3] = dX.d[3];
	m_memResult[idx].x.d[4] = dX.d[4];
	m_memResult[idx].x.d[5] = dX.d[5];
	m_memResult[idx].x.d[6] = dX.d[6];
	m_memResult[idx].x.d[7] = dX.d[7];
	m_memResult[idx].y.d[0] = tmp.d[0];
	m_memResult[idx].y.d[1] = tmp.d[1];
	m_memResult[idx].y.d[2] = tmp.d[2];
	m_memResult[idx].y.d[3] = tmp.d[3];
	m_memResult[idx].y.d[4] = tmp.d[4];
	m_memResult[idx].y.d[5] = tmp.d[5];
	m_memResult[idx].y.d[6] = tmp.d[6];
	m_memResult[idx].y.d[7] = tmp.d[7];
}

__global__ void dev_crack_search_gpu_cpu(mp_number* const pDeltaX, mp_number* const pInverse, mp_number* const pPrevLambda, const tableStruct* __restrict__ tables, point* __restrict__ m_memResultKeys, resultFound* __restrict__ m_memResult) {
	const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;;

	mp_number dX = pDeltaX[idx];
	mp_number tmp = pInverse[idx];
	mp_number lambda = pPrevLambda[idx];

	// λ' = - (2G_y) / d' - λ <=> lambda := pInversedNegativeDoubleGy[id] - pPrevLambda[id]
	mp_mod_sub(&lambda, &tmp, &lambda);

	// λ² = λ * λ <=> tmp := lambda * lambda = λ²
	mp_mod_mul(&tmp, &lambda, &lambda);

	// d' = λ² - d - 3g = (-3g) - (d - λ²) <=> x := tripleNegativeGx - (x - tmp)
	mp_mod_sub(&dX, &dX, &tmp);
	mp_mod_sub_const(&dX, &tripleNegativeGx, &dX);

	pDeltaX[idx] = dX;
	pPrevLambda[idx] = lambda;

	// Calculate y from dX and lambda
	// y' = (-G_Y) - λ * d' <=> p.y := negativeGy - (p.y * p.x)
	mp_mod_mul(&tmp, &lambda, &dX);
	mp_mod_sub_const(&tmp, &negativeGy, &tmp);

	// Restore X coordinate from delta value
	mp_mod_sub(&dX, &dX, &negativeGx);
	point pubKey;
	pubKey.x.d[0] = dX.d[0];
	pubKey.x.d[1] = dX.d[1];
	pubKey.x.d[2] = dX.d[2];
	pubKey.x.d[3] = dX.d[3];
	pubKey.x.d[4] = dX.d[4];
	pubKey.x.d[5] = dX.d[5];
	pubKey.x.d[6] = dX.d[6];
	pubKey.x.d[7] = dX.d[7];
	pubKey.y.d[0] = tmp.d[0];
	pubKey.y.d[1] = tmp.d[1];
	pubKey.y.d[2] = tmp.d[2];
	pubKey.y.d[3] = tmp.d[3];
	pubKey.y.d[4] = tmp.d[4];
	pubKey.y.d[5] = tmp.d[5];
	pubKey.y.d[6] = tmp.d[6];
	pubKey.y.d[7] = tmp.d[7];

	m_memResultKeys[idx].x.d[0] = dX.d[0];
	m_memResultKeys[idx].x.d[1] = dX.d[1];
	m_memResultKeys[idx].x.d[2] = dX.d[2];
	m_memResultKeys[idx].x.d[3] = dX.d[3];
	m_memResultKeys[idx].x.d[4] = dX.d[4];
	m_memResultKeys[idx].x.d[5] = dX.d[5];
	m_memResultKeys[idx].x.d[6] = dX.d[6];
	m_memResultKeys[idx].x.d[7] = dX.d[7];
	m_memResultKeys[idx].y.d[0] = tmp.d[0];
	m_memResultKeys[idx].y.d[1] = tmp.d[1];
	m_memResultKeys[idx].y.d[2] = tmp.d[2];
	m_memResultKeys[idx].y.d[3] = tmp.d[3];
	m_memResultKeys[idx].y.d[4] = tmp.d[4];
	m_memResultKeys[idx].y.d[5] = tmp.d[5];
	m_memResultKeys[idx].y.d[6] = tmp.d[6];
	m_memResultKeys[idx].y.d[7] = tmp.d[7];


	uint8_t num_table = *(uint8_t*)((uint8_t*)&pubKey.x.d[7] + 3);
	int ret = 0;
	uint64_t line_in_table_find;
	if (tables[num_table].size != 0)
	{
		const uint64_t bytes_y = *(const uint64_t*)&pubKey.x.d[6];
		ret = find_in_table_8_bytes(&pubKey, &tables[num_table], &line_in_table_find);
	}
	if (ret == 1)
	{
		m_memResult->score++;
		if (m_memResult->score >= NUM_RESULT_FOUND_KEYS)
		{
			m_memResult->score = 0;
		}
		uint32_t score = m_memResult->score;
		for (int ii = 0; ii < 8; ii++)
		{
			m_memResult->pub_key[score].x.d[ii] = pubKey.x.d[ii];
			m_memResult->pub_key[score].y.d[ii] = pubKey.y.d[ii];
		}
		m_memResult->line_in_tables[score] = line_in_table_find;
		m_memResult->id[score] = idx;
	}
}


